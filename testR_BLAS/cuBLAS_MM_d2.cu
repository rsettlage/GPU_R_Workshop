#include <R.h>
#include <stdio.h>
#include <cstdlib>
#include <hiprand.h>
#include <hipblas.h>
/* This function is written for R cuda matrix multiply.
going to use the hipblasDgemm
remember, hipblasDgemm is really prepping for a*(op)A %*% (op)B + b*C
NOT USED macro
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
*/

extern "C"
void cuMM(int *nr_A, int *nc_A, int *nc_B, double *A, double *B, double *C, double *a, double *b)
{
    // Set up variables
    const double alpha = (double) *a;
    const double beta = (double) *b;
   
    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
     
    // Allocate 3 arrays on GPU
    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A,*nr_A * *nc_A * sizeof(double));
    hipMalloc(&d_B,*nc_A * *nc_B * sizeof(double));
    hipMalloc(&d_C,*nr_A * *nc_B * sizeof(double));

    // Copy CPU data to GPU (could also use Unified Memory, beyond todays scope)
    hipMemcpy(d_A, A, *nr_A * *nc_A * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, *nc_A * *nc_B * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, *nr_A * *nc_B * sizeof(double), hipMemcpyHostToDevice);

    // Multiply A and B on GPU
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, *nr_A, *nc_B, *nc_A, &alpha, d_A, *nr_A, d_B, *nc_A, &beta, d_C, *nr_A);
    // Copy the data back to CPU
    hipMemcpy(C,d_C,*nr_A * *nc_B * sizeof(double),hipMemcpyDeviceToHost);
    
    //Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);  

    //return 0;
}
