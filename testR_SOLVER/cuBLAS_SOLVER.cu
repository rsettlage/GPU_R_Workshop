#include <R.h>
#include <stdio.h>
#include <cstdlib>
#include <hiprand.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
/* This function is written for R cuda matrix multiply followed by solve.
going to use the hipblasDgemm
remember, hipblasDgemm is really prepping for a*(op)A %*% (op)B + b*C
NOT USED macro
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
*/

extern "C"
void cuMM(int *nr_A, int *nc_A, int *nc_B, double *A, double *B, double *C, double *X, double *a, double *b)
{
    // Set up variables
    const double alpha = (double) *a;
    const double beta = (double) *b;
    int *d_Ipiv = NULL; /* pivoting sequence */ 
    int lwork = 0; /* size of workspace  */
    double *d_work = NULL; /* device workspace for getrf */
    int *d_info;

    // Create a handle for CUBLAS
    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);
    hipsolverHandle_t cusolverH;
    hipsolverDnCreate(&cusolverH);

    //printf("%d\n",*nc_B);
    // Allocate 4 arrays on GPU
    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A,*nr_A * *nc_A * sizeof(double));
    hipMalloc(&d_B,*nc_A * *nc_B * sizeof(double));
    hipMalloc(&d_C,*nr_A * *nc_A * sizeof(double));
    hipMalloc((void**)&d_Ipiv, sizeof(int) * *nr_A);
    hipMalloc((void**)&d_info, sizeof(int));

    // for solver, need to set aside some memory space and put it on device
    hipsolverDnDgetrf_bufferSize(cusolverH, *nr_A, *nc_A, d_A, *nr_A, &lwork);
    //hipsolverDnDpotrf_bufferSize(cusolverH, HIPBLAS_FILL_MODE_LOWER, *nr_A, d_A, *nr_A, &lwork);
    hipMalloc((void**)&d_work, sizeof(double)*lwork);

    // Copy CPU data to GPU (could also use Unified Memory, beyond todays scope)
    hipMemcpy(d_A, A, *nr_A * *nc_A * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, *nr_A * *nc_B * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, *nr_A * *nc_A * sizeof(double), hipMemcpyHostToDevice);

    // Compute (A'A)=C
    hipblasDgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, *nr_A, *nc_A, *nc_A, &alpha, d_A, *nr_A, d_A, *nc_A, &beta, d_C, *nr_A);
    // get C back, it gets over written when solving    
    hipMemcpy(C,d_C,*nr_A * *nc_A * sizeof(double),hipMemcpyDeviceToHost);

    // solve for X in CX=B where B is identity matrix
    hipsolverDnDgetrf(cusolverH, *nr_A, *nc_A, d_C, *nr_A, d_work, d_Ipiv, d_info);
    hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, *nr_A, *nc_B, d_C, *nr_A, d_Ipiv, d_B, *nc_B, d_info);
    //hipsolverDnDpotrf(cusolverH, HIPBLAS_FILL_MODE_LOWER, *nr_A, d_C, *nr_A, d_work, lwork, d_info);

    // Copy the X back to CPU (note that it is in d_B because solve overwrites it
    hipMemcpy(X,d_B,*nr_A * *nc_B * sizeof(double),hipMemcpyDeviceToHost);
    
    //Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);  
    if (cublasH) hipblasDestroy(cublasH);
    if (cusolverH) hipsolverDnDestroy(cusolverH);
    hipDeviceReset();
    //return 0;
}
